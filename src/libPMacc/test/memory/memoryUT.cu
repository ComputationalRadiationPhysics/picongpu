#include "hip/hip_runtime.h"
/**
 * Copyright 2015 Erik Zenker
 *
 * This file is part of libPMacc.
 *
 * libPMacc is free software: you can redistribute it and/or modify
 * it under the terms of either the GNU General Public License or
 * the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * libPMacc is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License and the GNU Lesser General Public License
 * for more details.
 *
 * You should have received a copy of the GNU General Public License
 * and the GNU Lesser General Public License along with libPMacc.
 * If not, see <http://www.gnu.org/licenses/>.
 */

// STL
#include <stdint.h> /* uint8_t */
#include <iostream> /* cout, endl */
#include <string>

// BOOST
#include <boost/test/unit_test.hpp>
#include <boost/mpl/list.hpp>
#include <boost/mpl/for_each.hpp>
#include <boost/mpl/integral_c.hpp>

// MPI
#include <mpi.h> /* MPI_Init, MPI_Finalize */

// PMacc
#include <Environment.hpp>
#include <dimensions/DataSpace.hpp>
#include <memory/buffers/HostBufferIntern.hpp>
#include <memory/buffers/HostBuffer.hpp>
#include <memory/buffers/DeviceBufferIntern.hpp>
#include <memory/buffers/DeviceBuffer.hpp>
#include <dimensions/DataSpace.hpp>
#include <types.h> /* DIM1,DIM2,DIM3 */


/*******************************************************************************
 * Configuration
 ******************************************************************************/

/**
 * A fixture is an object that is constructed before some
 * statment and destructed after some statement. Thus, the
 * fixture defines pre and postconditions of this statement.
 *
 * This fixture defines the initialization and termination
 * of MPI and the initialization of the environment
 * singleton.
 */
struct Fixture {
    Fixture(){
        int argc = 0;
        char **argv = NULL;

        MPI_Init( &argc, &argv );

        PMacc::DataSpace<DIM3> const devices(1,1,1);
        PMacc::DataSpace<DIM3> const periodic(1,1,1);
        PMacc::Environment<DIM3>::get().initDevices(devices, periodic);


    }

    ~Fixture(){
        MPI_Finalize( );
    }

};


/**
 * Defines for which numbers of elements a
 * test should be verfied e.g. the size
 * of a host or device buffer.
 */
template<typename T_Dim>
std::vector<size_t> getElementsPerDim(){
    std::vector<size_t> nElements;
    std::vector<size_t> nElementsPerDim;

    // Elements total
    nElements.push_back(1);
    nElements.push_back(1 * 1000);
    nElements.push_back(1 * 1000 * 1000);
    nElements.push_back(1 * 1000 * 1000 * 10);

    // Elements per dimension
    for(size_t i = 0; i < nElements.size(); ++i){
        nElementsPerDim.push_back(std::pow(nElements[i], static_cast<double>(1)/static_cast<double>(T_Dim::value))); 

    }
    return nElementsPerDim;
}


/**
 * Definition of a list of dimension types. This
 * List is used to test memory operations in
 * each dimension setup automatically. For this
 * purpose boost::mpl::for_each is used.
 */
typedef ::boost::mpl::list<boost::mpl::integral_c<int, DIM1>,
                           boost::mpl::integral_c<int, DIM2>,
                           boost::mpl::integral_c<int, DIM3> > Dims;

BOOST_GLOBAL_FIXTURE( Fixture );


/*******************************************************************************
 * Test Suites
 ******************************************************************************/
BOOST_AUTO_TEST_SUITE( memory )

  BOOST_AUTO_TEST_SUITE( HostBufferIntern )
  #include "HostBufferIntern/getPointer.hpp"
  #include "HostBufferIntern/reset.hpp"
  #include "HostBufferIntern/copyFrom.hpp"
  BOOST_AUTO_TEST_SUITE_END()

BOOST_AUTO_TEST_SUITE_END()
