#include "hip/hip_runtime.h"
/*
 * Illinois Open Source License
 *
 * University of Illinois/NCSA
 * Open Source License
 *
 * Copyright � 2009,    University of Illinois.  All rights reserved.
 *
 * Developed by:
 *
 * Innovative Systems Lab
 * National Center for Supercomputing Applications
 * http://www.ncsa.uiuc.edu/AboutUs/Directorates/ISL.html
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the "Software"), to deal with
 * the Software without restriction, including without limitation the rights to use,
 * copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the
 * Software, and to permit persons to whom the Software is furnished to do so, subject
 * to the following conditions:
 *
 * * Redistributions of source code must retain the above copyright notice, this list
 * of conditions and the following disclaimers.
 *
 * * Redistributions in binary form must reproduce the above copyright notice, this list
 * of conditions and the following disclaimers in the documentation and/or other materials
 * provided with the distribution.
 *
 * * Neither the names of the Innovative Systems Lab, the National Center for Supercomputing
 * Applications, nor the names of its contributors may be used to endorse or promote products
 * derived from this Software without specific prior written permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
 * INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR
 * PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE
 * LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT
 * OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS WITH THE SOFTWARE.
 */

#include "cuda_memtest.h"
#include <hipblas.h>
#include <pthread.h>
#include <stdio.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/time.h>
#include <signal.h>
#include <hip/hip_runtime.h>

#define MAX_NUM_GPUS 8
bool useMappedMemory;
void* mappedHostPtr;
char hostname[64];
unsigned int verbose =0;
unsigned int interactive =0;
extern cuda_memtest_t cuda_memtests[11];
unsigned int max_num_blocks = 0;
unsigned int exit_on_error = 0;
unsigned int monitor_temp = 0;
unsigned int monitor_interval = 5;
unsigned int email_notification = 0;
unsigned int global_pattern = 0;
unsigned long global_pattern_long = 0;
char emails[128];
unsigned int report_interval = 1800;  //senconds
unsigned long long serial_number = 0;
unsigned int num_iterations = 1000;
unsigned int num_passes = 0;
unsigned int healthy_threads = 0;
unsigned int disable_serial_number = 0;
__thread unsigned int gpu_idx;
char driver_info[MAX_STR_LEN];

pthread_mutex_t mutex = PTHREAD_MUTEX_INITIALIZER;
pthread_mutex_t atomic_mutex = PTHREAD_MUTEX_INITIALIZER;

void run_tests(char*, unsigned int);
extern void update_temperature(void);
extern unsigned long long get_serial_number(void);
extern void allocate_small_mem(void);

typedef struct arg_s{
    unsigned int device;
}arg_t;

/*

struct hipDeviceProp_t {
    char name[256];
    size_t totalGlobalMem;
    size_t sharedMemPerBlock;
    int regsPerBlock;
    int warpSize;
    size_t memPitch;
    int maxThreadsPerBlock;
    int maxThreadsDim[3];
    int maxGridSize[3];
    size_t totalConstMem;
    int major;
    int minor;
    int clockRate;
    size_t textureAlignment;
    int deviceOverlap;
    int multiProcessorCount;
    int kernelExecTimeoutEnabled;
}
*/
void
display_device_info(struct hipDeviceProp_t* prop)
{
    PRINTF("Device name=%s, global memory size=%llu\n", prop->name, prop->totalGlobalMem);
    return;
}


void
atomic_inc(unsigned int* value)
{
    pthread_mutex_lock(&atomic_mutex);
    (*value)= (*value) + 1;
    pthread_mutex_unlock(&atomic_mutex);
}

unsigned int
atomic_read(unsigned int* value)
{
    unsigned int ret;

    pthread_mutex_lock(&atomic_mutex);
    ret = *value;
    pthread_mutex_unlock(&atomic_mutex);

    return ret;
}

void*
thread_func(void* _arg)
{

    arg_t* arg = (arg_t*)_arg;
    unsigned int device = arg->device;
    gpu_idx = device;



    struct hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device); CUERR;

    display_device_info(&prop);

    unsigned long totmem = prop.totalGlobalMem;

    PRINTF("major=%d, minor=%d\n", prop.major, prop.minor);

    //need to leave a little headroom or later calls will fail
    unsigned int tot_num_blocks = totmem/BLOCKSIZE -16;
    if (max_num_blocks != 0){
	tot_num_blocks = MIN(max_num_blocks+16, tot_num_blocks);
    }


    hipSetDevice(device);
    hipDeviceSynchronize();
    CUERR;

    PRINTF("Attached to device %d successfully.\n", device);

    size_t free, total;
    hipMemGetInfo(&free, &total);

    allocate_small_mem();

    char* ptr = NULL;

    tot_num_blocks = MIN(tot_num_blocks, free/BLOCKSIZE - 16);
    do{
        tot_num_blocks -= 16 ; //magic number 16 MB
        DEBUG_PRINTF("Trying to allocate %d MB\n", tot_num_blocks);
        if (tot_num_blocks <= 0){
            FPRINTF("ERROR: cannot allocate any memory from GPU\n");
            exit(ERR_GENERAL);
        }
        if(useMappedMemory)
        {
            //create cuda mapped memory
            hipHostAlloc((void**)&mappedHostPtr,tot_num_blocks* BLOCKSIZE,hipHostMallocMapped);
            hipHostGetDevicePointer(&ptr,mappedHostPtr,0);
        }
        else
        {
            hipMalloc((void**)&ptr, tot_num_blocks* BLOCKSIZE);
        }
    }while(hipGetLastError() != hipSuccess);

    PRINTF("Allocated %d MB\n", tot_num_blocks);

    atomic_inc(&healthy_threads);
    run_tests(ptr, tot_num_blocks);

    return NULL;

}


void*
temp_monitor_thread_func(void* arg)
{
    do{
	update_temperature();
	sleep(monitor_interval);
    }while(1);

}


void list_tests_info(void)
{
    int i;
    for (i = 0;i < DIM(cuda_memtests); i++){
	printf("%s %s\n", cuda_memtests[i].desc, cuda_memtests[i].enabled?"":" ==disabled by default==");
    }
    return;
}


void
usage(char** argv)
{

    char example_usage[] =
	"run on default setting:       ./cuda_memtest\n"
	"run on stress test only:      ./cuda_memtest --stress\n";

    printf("Usage:%s [options]\n", argv[0]);
    printf("options:\n");
    printf("--mappedMem                 run all checks with cuda mapped memory instead of native device memory\n");
    printf("--silent                    Do not print out progress message (default)\n");
    printf("--device <idx>              Designate one device for test\n");
    printf("--interactive               Progress info will be printed in the same line\n");
    printf("--disable_all               Disable all tests\n");
    printf("--enable_test <test_idx>    Enable the test <test_idx>\n");
    printf("--disable_test <test_idx>   Disable the test <test_idx>\n");
    printf("--max_num_blocks <n>        Set the maximum of blocks of memory to test\n");
    printf("                            1 block = 1 MB in here\n");
    printf("--exit_on_error             When finding error, print error message and exit\n");
    printf("--monitor_temp <interval>   Monitoring temperature, the temperature will be updated every <interval> seconds\n");
    printf("                            This feature is experimental\n");
    printf("--emails <a@b,c@d,...>      Setting email notification\n");
    printf("--report_interval <n>       Setting the interval in seconds between email notifications(default 1800)\n");
    printf("--pattern <pattern>         Manually set test pattern for test4/test8/test10\n");
    printf("--list_tests                List all test descriptions\n");
    printf("--num_iterations <n>        Set the number of iterations (only effective on test0 and test10)\n");
    printf("--num_passes <n>            Set the number of test passes (this affects all tests)\n");
    printf("--disable_serial_number     Disable reporting serial number\n");
    printf("--verbose <n>               Setting verbose level\n");
    printf("                              0 -- Print out test start and end message only (default)\n");
    printf("                              1 -- Print out pattern messages in test\n");
    printf("                              2 -- Print out progress messages\n");
    printf("--stress                    Stress test. Equivalent to --disable_all --enable_test 10 --exit_on_error\n");
    printf("--help                      Print this message\n");
    printf("\nExample usage:\n\n");
    printf("%s\n", example_usage);

    exit(ERR_GENERAL);
}


int
main(int argc, char** argv)
{
    int i;
    useMappedMemory=false;
    mappedHostPtr=NULL;

    if (argc >=2 ){
	if( strcmp(argv[1], "--help")== 0){
	    usage(argv);
	}
    }

    if(gethostname(hostname, 64) !=0){
	fprintf(stderr, "ERROR: gethostname() returns error\n");
	exit(ERR_GENERAL);
    }

    for(i=0;i < 64; i++){
	if (hostname[i] == '.'){
	    hostname[i] = 0;
	    break;
	}
    }

    PRINTF("Running cuda memtest, version %s\n", VERSION);
    int device = -1;
    int num_gpus;
    hipGetDeviceCount(&num_gpus);CUERR;

    if (num_gpus == 0){
	fprintf(stderr,"ERROR: no GPUs found\n");
	exit(ERR_GENERAL);
    }


    for (i =1;i < argc; i++){

	if( strcmp(argv[i], "--help")== 0){
	    usage(argv);
	}

    if( strcmp(argv[i], "--mappedMem")== 0){
	    useMappedMemory=true;
        continue;
	}

	if( strcmp(argv[i], "--verbose") == 0){
	    if (i+1 >= argc){
		usage(argv);
	    }
	    verbose = atoi(argv[i+1]);
	    i++;
	    continue;

	}
	if (strcmp(argv[i], "--silent") == 0){
	    verbose = 0;
	    continue;
	}
	if (strcmp(argv[i], "--interactive") == 0){
	    interactive = 1;
	    continue;
	}
	if (strcmp(argv[i], "--noninteractive") == 0){
	    interactive = 0;
	    continue;
	}
	if (strcmp(argv[i], "--enable_test") == 0){
	    if (i+1 >= argc){
		usage(argv);
	    }
	    int idx = atoi(argv[i+1]);
	    if (idx >= DIM(cuda_memtests)){
		fprintf(stderr, "Error: invalid test id\n");
		usage(argv);
	    }

	    cuda_memtests[idx].enabled = 1;

	    i++;
	    continue;
	}
	if (strcmp(argv[i], "--disable_test") == 0){
	    if (i+1 >= argc){
		usage(argv);
	    }
	    int idx = atoi(argv[i+1]);
	    if (idx >= DIM(cuda_memtests)){
		fprintf(stderr, "Error: invalid test id\n");
		usage(argv);
	    }

	    cuda_memtests[idx].enabled = 0;
	    i++;
	    continue;
	}
	if (strcmp(argv[i], "--disable_all") == 0){
	    int k;
	    for (k=0;k < DIM(cuda_memtests);k++){
		cuda_memtests[k].enabled = 0;
	    }
	    continue;
	}

	if (strcmp(argv[i], "--device") == 0){
	    if (i+1 >= argc){
		usage(argv);
	    }
	    device = atoi(argv[i+1]);
	    i++;
	    num_gpus = 1;
	    continue;
	}

	if (strcmp(argv[i], "--max_num_blocks") == 0){
	    if (i+1 >= argc){
		usage(argv);
	    }
	    max_num_blocks = atoi(argv[i+1]);
	    i++;
	    continue;
	}

	if (strcmp(argv[i], "--exit_on_error") == 0){
	    exit_on_error = 1;
	    continue;
	}

	if (strcmp(argv[i], "--monitor_temp") == 0){
	    monitor_temp =1;
	    if (i+1 >= argc){
		usage(argv);
	    }
	    monitor_interval = atoi(argv[i+1]);
	    i++;
	    continue;
	}
	if (strcmp(argv[i], "--pattern") == 0){
	    if (i+1 >= argc){
		usage(argv);
	    }
	    sscanf(argv[i+1], "0x%lx", &global_pattern_long);
	    if (global_pattern_long ==0){
		printf("ERROR: global test pattern cannot be zero\n");
		usage(argv);
	    }
	    printf("Using global test pattern: 0x%lx\n", global_pattern_long);
	    global_pattern = (unsigned long)global_pattern_long;
	    i++;
	    continue;
	}
	if (strcmp(argv[i], "--emails") == 0){
	    email_notification =1;

	    struct stat statbuf;
	    if (stat(MAILFILE, &statbuf)!=0){
		fprintf(stderr, "ERROR: stating mail unitility(%s) failed\n", MAILFILE);
		usage(argv);
	    }

	    if( !(S_IXOTH & statbuf.st_mode)){
		fprintf(stderr, "ERROR: no permission on exeution on the mail utility\n");
		usage(argv);
	    }


	    if (i+1 >= argc){
		usage(argv);
	    }
	    if ( strlen( argv[i+1]) > sizeof(emails)){
		fprintf(stderr, "ERROR: email string too long\n");
		usage(argv);
	    }
	    strcpy(emails, argv[i+1]);
	    i++;
	    continue;
	}
	if (strcmp(argv[i], "--report_interval") == 0){

	    if (i+1 >= argc){
		usage(argv);
	    }
	    report_interval = atoi(argv[i+1]);
	    i++;
	    continue;
	}

	if (strcmp(argv[i], "--num_iterations") == 0){

	    if (i+1 >= argc){
		usage(argv);
	    }
	    num_iterations = atoi(argv[i+1]);
	    if (num_iterations <= 0){
		printf("ERROR: invalid number of iterations\n");
		usage(argv);
	    }
	    i++;
	    continue;
	}

	if (strcmp(argv[i], "--num_passes") == 0){

	    if (i+1 >= argc){
		usage(argv);
	    }
	    num_passes = atoi(argv[i+1]);
	    if (num_passes <= 0){
		printf("ERROR: invalid number of passes\n");
		usage(argv);
	    }
	    i++;
	    continue;
	}

	if (strcmp(argv[i], "--disable_serial_number") == 0){
	    disable_serial_number= 1;
	    continue;
	}

	if (strcmp(argv[i], "--stress") == 0){
	    //equal to "--disable_all --enable_test 10 --exit_on_error"
	    int k;
	    for (k=0;k < DIM(cuda_memtests);k++){
		cuda_memtests[k].enabled = 0;
	    }
	    cuda_memtests[10].enabled = 1;
	    exit_on_error = 1;
	    continue;
	}

	if (strcmp(argv[i], "--list_tests") == 0){
	    list_tests_info();
	    return 0;
	}
	fprintf(stderr, "ERROR: Invalid option:%s\n", argv[i]);
	usage(argv);
    }

    if (!disable_serial_number){
	serial_number  = get_serial_number();
    }

    get_driver_info(driver_info, MAX_STR_LEN);

    PRINTF("num_gpus=%d\n", num_gpus);
    if(num_gpus > MAX_NUM_GPUS){
	fprintf(stderr, "Error: max number of GPUs (%d) exceeded: %d\n", MAX_NUM_GPUS, num_gpus);
    }
    pthread_t temp_pid;
    if (monitor_temp){
	if (pthread_create(&temp_pid, NULL, temp_monitor_thread_func, NULL)  != 0){
	    printf("ERROR: creating thread for temperature monitoring failed\n");
	    exit(ERR_GENERAL);
	}
    }

    arg_t args[MAX_NUM_GPUS];
    pthread_t pid[MAX_NUM_GPUS];

    if (device != -1){ //device set, only 1 GPU
	args[0].device = device;
	pthread_create(&pid[0], NULL, thread_func, (void*)&args[0]);
    }else{//multiple GPUs
	for (i=0;i < num_gpus;i++){
	    args[i].device = i;
	    pthread_create(&pid[i], NULL, thread_func, (void*)&args[i]);
	}

    }

    struct timeval t0, t1;
    int ht=0;
    double wait_time = 500;
    gettimeofday(&t0, NULL);

    while(1){
	ht = atomic_read(&healthy_threads);
	if (ht == num_gpus){
	    break;
	}

	gettimeofday(&t1, NULL);
	double passed_time = TDIFF(t1, t0);
	if (passed_time >= wait_time){
	    break;
	}
	sleep(1);
    }

    if (ht < num_gpus){
	printf("ERROR: Some GPU threads are not progressing (healthy_threads=%d, num_gpus=%d)\n", ht, num_gpus);
	fflush(stdout); fflush(stderr);
	for(i=0;i < num_gpus;i++){
		pthread_kill(pid[i], SIGTERM);
	}
	exit(ERR_BAD_STATE);
    }


    for(i=0;i < num_gpus;i++){
	pthread_join(pid[i], NULL);
    }

    printf("main thread: Program exits\n");

    return 0;
}

